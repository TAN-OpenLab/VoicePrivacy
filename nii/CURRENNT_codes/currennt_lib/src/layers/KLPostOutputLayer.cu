#include "hip/hip_runtime.h"
/******************************************************************************
 * This file is an addtional component of CURRENNT. 
 * Xin WANG
 * National Institute of Informatics, Japan
 * 2016
 *
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "KLPostOutputLayer.hpp"

#include "../Configuration.hpp"
#include "../helpers/JsonClasses.hpp"
#include "../helpers/getRawPointer.cuh"
#include "../helpers/NumericLimits.cuh"
#include "../helpers/min.cuh"
#include "../helpers/max.cuh"

#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#define KLDOUTPUTDATATYPE_LINEAR_UNI 1
#define KLDOUTPUTDATATYPE_LOG_UNI 2
#include <cmath>

namespace internal {
namespace {

    
    // For logData
    struct ComputeKLDFnLog
    {
        int layerSize;
	//int maxTime;
	
        const char   *patTypes;
	const real_t *mvData;
	real_t       *errorBuf;
	real_t        factor;
        __host__ __device__ real_t operator() (const thrust::tuple<real_t, real_t, int> &values) const
        {
            // unpack the tuple
            real_t target = values.get<0>();
            real_t output = values.get<1>();
            int outputIdx = values.get<2>();
	    
            // check if we have to skip this value
            int patIdx = outputIdx    / layerSize;
            if (patTypes[patIdx] == PATTYPE_NONE){
		*(errorBuf + outputIdx) = 0;
                return 0;
	    }
	    
	    int featDim   = outputIdx % layerSize;
	    
	    // get linear value
	    real_t mean   = *(mvData + featDim);
	    real_t var    = *(mvData + featDim + layerSize);
	    real_t tarLin = var*target + mean;
	    real_t outLin = var*output + mean;
	    
	    // upper unbounded
	    tarLin = (tarLin > -helpers::NumericLimits<real_t>::expLimit())?(exp(tarLin)):(0);
	    outLin = (outLin > -helpers::NumericLimits<real_t>::expLimit())?(exp(outLin)):(0);
	    
	    *(errorBuf + outputIdx) = (var * (outLin - tarLin))/factor;
            return (tarLin * (var * (target - output) - 1) + outLin)/factor;
        }
    };

    // For linear Data
    struct ComputeKLDFnLinear
    {
        int layerSize;
	//int maxTime;
	
        const char   *patTypes;
	const real_t *mvData;
	real_t       *errorBuf;
	real_t        factor;
        __host__ __device__ real_t operator() (const thrust::tuple<real_t, real_t, int> &values) const
        {
            // unpack the tuple
            real_t target = values.get<0>();
            real_t output = values.get<1>();
            int outputIdx = values.get<2>();
	    
            // check if we have to skip this value
            int patIdx = outputIdx    / layerSize;
            if (patTypes[patIdx] == PATTYPE_NONE){
		*(errorBuf + outputIdx) = 0;
                return 0;
	    }
	    
	    int featDim   = outputIdx % layerSize;
	    
	    // get linear value
	    real_t mean   = *(mvData + featDim);
	    real_t var    = *(mvData + featDim + layerSize);
	    real_t tarLin = var*target + mean;
	    real_t outLin = var*output + mean;
	    
	    // upper unbounded
	    real_t ratio = helpers::max(helpers::NumericLimits<real_t>::min(), tarLin/outLin);
	    
	    *(errorBuf + outputIdx) = (var * (1 - ratio))/factor;
	    
            return (tarLin * log(ratio) - tarLin + outLin)/factor;
        }
    };
    
    
} // anonymous namespace
} // namespace anonymous


namespace layers {

    template <typename TDevice>
    KLPostOutputLayer<TDevice>::KLPostOutputLayer(const helpers::JsonValue &layerChild, 
						  Layer<TDevice> &precedingLayer,
						  int             maxSeqLength,
						  int             layerID)
        : PostOutputLayer<TDevice>(layerChild, precedingLayer,
				   precedingLayer.size(), maxSeqLength, layerID)
    {
	/* Load options */
	const Configuration &config = Configuration::instance();
	m_dataType = config.KLDOutputDataType();
	
	if (m_dataType != KLDOUTPUTDATATYPE_LINEAR_UNI && m_dataType != KLDOUTPUTDATATYPE_LOG_UNI){
	    // Linear-domain spectrum vector or log domain?
	    m_dataType = (layerChild->HasMember("kld_log_or_linear") ? 
			  ((*layerChild)["kld_log_or_linear"].GetInt()) : KLDOUTPUTDATATYPE_LOG_UNI);
	}
	
	if (m_dataType == KLDOUTPUTDATATYPE_LOG_UNI)
	    printf("\n\tTarget Data type : log data");
	else
	    printf("\n\tTarget Data type : linear data");
	
	m_lrFactor = config.lrFactor();
	printf("\n\tKLD scale factor: %f", m_lrFactor);

	// initialize the buffer
	m_errorBuf = this->_actualOutputs();
	
    }

    template <typename TDevice>
    KLPostOutputLayer<TDevice>::~KLPostOutputLayer()
    {
    }

    template <typename TDevice>
    const std::string& KLPostOutputLayer<TDevice>::type() const
    {
        static const std::string s("kld");
        return s;
    }

    template <typename TDevice>
    real_t KLPostOutputLayer<TDevice>::calculateError()
    {
		    
	if(m_dataType == KLDOUTPUTDATATYPE_LINEAR_UNI){
	    internal::ComputeKLDFnLinear fn;
	    fn.layerSize = this->size();
	    fn.patTypes  = helpers::getRawPointer(this->patTypes());
	    fn.mvData    = helpers::getRawPointer(this->_mvVector());
	    fn.errorBuf  = helpers::getRawPointer(m_errorBuf);
	    fn.factor    = m_lrFactor;
	    
	    //fn.maxTime   = this->maxSeqLength();
	    int n = this->curMaxSeqLength() * this->parallelSequences() * this->size();

	    real_t mse = (real_t) thrust::transform_reduce(
               thrust::make_zip_iterator(
		   thrust::make_tuple(this->_targets().begin(),   
				      this->_actualOutputs().begin(),   
				      thrust::counting_iterator<int>(0))),
               thrust::make_zip_iterator(
		   thrust::make_tuple(this->_targets().begin()+n, 
				      this->_actualOutputs().begin()+n, 
				      thrust::counting_iterator<int>(0)+n)),
	       fn,
	       (real_t)0,
	       thrust::plus<real_t>()
	       );
	    
	    return mse;
	    
	}else if(m_dataType == KLDOUTPUTDATATYPE_LOG_UNI){
	    
	    internal::ComputeKLDFnLog fn;
	    fn.layerSize = this->size();
	    fn.patTypes  = helpers::getRawPointer(this->patTypes());
	    fn.mvData    = helpers::getRawPointer(this->_mvVector());
	    fn.errorBuf  = helpers::getRawPointer(m_errorBuf);
	    fn.factor    = m_lrFactor;
	    
	    //fn.maxTime   = this->maxSeqLength();
	    int n = this->curMaxSeqLength() * this->parallelSequences() * this->size();

	    real_t mse = (real_t) thrust::transform_reduce(
               thrust::make_zip_iterator(
		   thrust::make_tuple(this->_targets().begin(),   
				      this->_actualOutputs().begin(),   
				      thrust::counting_iterator<int>(0))),
               thrust::make_zip_iterator(
		   thrust::make_tuple(this->_targets().begin()+n, 
				      this->_actualOutputs().begin()+n, 
				      thrust::counting_iterator<int>(0)+n)),
	       fn,
	       (real_t)0,
	       thrust::plus<real_t>()
	       );
	    return mse;
	    
	}else{
	    throw std::runtime_error(std::string("Error: kld_log_or_linear: 1 or 2 in network.jsn"));
	}
    }

    template <typename TDevice>
    void KLPostOutputLayer<TDevice>::computeForwardPass(const int nnState)
    {
    }

    template <typename TDevice>
    void KLPostOutputLayer<TDevice>::computeForwardPass(const int timeStep, const int nnState)
    {
    }

    template <typename TDevice>
    void KLPostOutputLayer<TDevice>::computeBackwardPass(const int nnState)
    {
     // calculate the errors
	/*internal::ComputeOutputErrorFn fn;
	  fn.layerSize = this->size();
	  fn.patTypes  = helpers::getRawPointer(this->patTypes());
	  fn.mvData    = helpers::getRawPointer(this->_mvVector());*/
	
	int n = this->curMaxSeqLength() * this->parallelSequences() * this->size();
	
	/*thrust::transform(
	  thrust::make_zip_iterator(
	  thrust::make_tuple(this->_actualOutputs().begin(),   
	  this->_targets().begin(),   
	  thrust::counting_iterator<int>(0))),
	  thrust::make_zip_iterator(
	  thrust::make_tuple(this->_actualOutputs().begin()+n, 
	  this->_targets().begin()+n, 
	  thrust::counting_iterator<int>(0)+n)),
	  this->_outputErrors().begin(),
	  fn
	  );*/
	
	thrust::copy(m_errorBuf.begin(), m_errorBuf.begin() + n, this->_outputErrors().begin());
    }

    template <typename TDevice>
    void KLPostOutputLayer<TDevice>::computeBackwardPass(const int timeStep, const int nnState)
    {
	if (timeStep == this->curMaxSeqLength())
	    this->computeBackwardPass(nnState);
    }


    template <typename TDevice>
    void KLPostOutputLayer<TDevice>::exportLayer(const helpers::JsonValue     &layersArray, 
						     const helpers::JsonAllocator &allocator) const
    {
        PostOutputLayer<TDevice>::exportLayer(layersArray, allocator);
	(*layersArray)[layersArray->Size() - 1].AddMember("kld_log_or_linear", m_dataType, allocator);
    }


    
    // explicit template instantiations
    template class KLPostOutputLayer<Cpu>;
    template class KLPostOutputLayer<Gpu>;

} // namespace layers
