#include "hip/hip_runtime.h"
/******************************************************************************
 * This file is an addtional component of CURRENNT. 
 * Xin WANG
 * National Institute of Informatics, Japan
 * 2019
 *
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "SelfAttention.hpp"
#include "../helpers/getRawPointer.cuh"
#include "../helpers/Matrix.hpp"
#include "../helpers/min.cuh"
#include "../helpers/max.cuh"
#include "../helpers/safeExp.cuh"

#include "../helpers/misFuncs.hpp"
#include "../helpers/JsonClasses.hpp"
#include "../Configuration.hpp"
#include "../MacroDefine.hpp"
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <math.h> 
#include <typeinfo>

#define PI_DEFINITION 3.141592653589793f

namespace internal {
namespace {
    struct CalculateOffsetFn
    {
        int layerSize;

        const real_t *outputs;

        __host__ __device__ real_t operator() (const int &patIdx) const
        {

            // search for the min and max output
            real_t max = helpers::NumericLimits<real_t>::min();
            real_t min = helpers::NumericLimits<real_t>::max();

            const real_t *offOutputs = &outputs[patIdx * layerSize];

            for (int i = 0; i < layerSize; ++i) {
                real_t x = offOutputs[i];
                min = helpers::min(min, x);
                max = helpers::max(max, x);
            }

            // calculate the offset
            real_t offset = (real_t)0.5 * (min + max);

            return offset;
        }
    };

    struct CalculateExpFn
    {
        int    layerSize;
	int    epoch;
	real_t prior_w;
	
        const real_t *offsets;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
        {
            // unpack the tuple
            real_t output = t.get<0>();

            // calculate the pattern index
            int outputIdx = t.get<1>() / layerSize;
	    int inputIdx  = t.get<1>() % layerSize;
	    
            // check if we can stop the calculation
            real_t offset = offsets[outputIdx];

	    // prior weight: use Gaussian window * decay_factor * relative_amplitude
	    real_t prior = helpers::safeExp(-1.0 * (outputIdx-inputIdx) * (outputIdx-inputIdx)/5.0) * 
		powf(prior_w, epoch) * fabsf(offset);
		
            // calculate the exponent
	    real_t x = helpers::safeExp(output - offset + prior);

            // store the result
            t.get<0>() = x;
        }
    };

    /*
    struct SumUpOutputsFn
    {
        int layerSize;

        const real_t *outputs;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
        {
            // unpack the tuple
            int patIdx = t.get<1>();

            // sum up the outputs
            const real_t *offOutputs = &outputs[patIdx * layerSize];

            real_t sum = 0;
            for (int i = 0; i < layerSize; ++i)
                sum += offOutputs[i];

            // store the result
            t.get<0>() = sum;
        }
	};*/

    struct NormalizeOutputsFn
    {
        int layerSize;

        const real_t *normFacts;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
        {
            // unpack the tuple
            int outputIdx = t.get<1>();

            // calculate the pattern index
            int patIdx = outputIdx / layerSize;

            // check if we can stop the calculation
            real_t normFact = normFacts[patIdx];

            // calculate the normalized value
            real_t x = t.get<0>() / normFact;

            // store the result
            t.get<0>() = x;
        }
    };

    struct CalculateErrorOffsetFn
    {
        int layerSize;

        const real_t *outputs;
        const real_t *outputErrors;

        __host__ __device__ real_t operator() (const int &patIdx) const
        {
            // calculate the offset
            const real_t *offOutputs      = &outputs     [patIdx * layerSize];
            const real_t *offOutputErrors = &outputErrors[patIdx * layerSize];

            real_t offset = 0;
            for (int i = 0; i < layerSize; ++i)
                offset += offOutputs[i] * offOutputErrors[i];

            return offset;
        }
    };
    

    struct CalculateErrorsFn
    {
        int layerSize;

        const real_t *errorOffsets;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, const real_t&, int> &t) const
        {
            // unpack the tuple
            int outputIdx = t.get<2>();

            // calculate the pattern index
            int patIdx = outputIdx / layerSize;

            // check if we can stop the calculation
            real_t offset = errorOffsets[patIdx];
    
            // calculate the delta
            real_t error  = t.get<0>();
            real_t output = t.get<1>();

            real_t x = output * (error - offset);

            // store the result
            t.get<0>() = x;
        }
    };

} // anonymous namespace
} // namespace internal


namespace layers {

    template <typename TDevice>
    SelfAttentionLayer<TDevice>::SelfAttentionLayer(const helpers::JsonValue &layerChild, 
						    const helpers::JsonValue &weightsSection, 
						    Layer<TDevice> &precedingLayer,
						    int             maxSeqLength,
						    int             layerID)
        : TrainableLayer<TDevice>(layerChild, weightsSection,
				  3, 0, precedingLayer, maxSeqLength, layerID)
    {
	if (this->parallelSequences() > 1)
	    throw std::runtime_error("Self-attention not implemented for parallel_seq > 1");
	this->__loadOpts(layerChild);
	this->__allocateLocalMem();
    }

    template <typename TDevice>
    SelfAttentionLayer<TDevice>::~SelfAttentionLayer()
    {
    }

    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::__loadOpts(const helpers::JsonValue &layerChild)
    {
	// 
	m_align_prior_w = (layerChild->HasMember("alignPriorWeight")?
			   static_cast<real_t>((*layerChild)["alignPriorWeight"].GetDouble()) :
			   0.99);
	

    }

    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::__allocateLocalMem()
    {

	Cpu::real_vector tmp;
	tmp.resize(this->outputs().size(), 0.0);

	m_mat_v = tmp;
	m_mat_k = tmp;
	m_mat_q = tmp;
	m_grad_buf = tmp;

	
	tmp.resize((this->outputs().size()/this->size()/this->parallelSequences()) *
		   (this->outputs().size()/this->size()),
		   0.0);
	m_align = tmp;
	m_align_grad = tmp;


	tmp.resize(this->outputs().size()/this->size(),0.0);
	m_softmax_buf = tmp;
	m_one_vector = tmp;
	thrust::fill(m_one_vector.begin(), m_one_vector.end(), 1.0);
    }

    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::__clearLocalMem()
    {
	m_mat_v.clear(); m_mat_v.shrink_to_fit();
	m_mat_k.clear(); m_mat_k.shrink_to_fit();	
	m_mat_q.clear(); m_mat_q.shrink_to_fit();
	
	m_grad_buf.clear(); m_grad_buf.shrink_to_fit();

	m_align.clear(); m_align.shrink_to_fit();
	m_align_grad.clear(); m_align_grad.shrink_to_fit();
	m_softmax_buf.clear(); m_softmax_buf.shrink_to_fit();
	
	m_one_vector.clear(); m_one_vector.shrink_to_fit();
	return;
    }

    template <typename TDevice>
    const std::string& SelfAttentionLayer<TDevice>::type() const
    {
        static const std::string s = "self_attention";
        return s;
    }

    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::computeForwardPass(const int nnState)
    {
	if (this->getSaveMemoryFlag())
	    throw std::runtime_error("Memory save mode should be turned off");

	// matrix size for W_v, W_q, and W_k
	int tmp_mat_size = this->precedingLayer().size() * this->size();

	// total number of time steps
	int frame_num_total = this->curMaxSeqLength() * this->parallelSequences();
	
	
	{{
	    // step1. compute Key, Query, Value
		
	    // x: input sequence of vectors
	    helpers::Matrix<TDevice> mat_pre_o(&this->precedingLayer().outputs(), 
					       this->precedingLayer().size(), 
					       frame_num_total);

	    // Value 
	    // mat_w_v: transformation matrix W_v
	    //  note: mat_w_v has dimension [input_size, output_size]
	    helpers::Matrix<TDevice> mat_w_v(&this->weights(),                  
					     this->precedingLayer().size(), this->size());
	    // mat_v: W_v^T * x
            helpers::Matrix<TDevice> mat_v(&m_mat_v, this->size(), frame_num_total);
            mat_v.assignProduct(mat_w_v, true, mat_pre_o, false);

	    // Query
	    // mat_w_q: transformation matrix W_q
	    helpers::Matrix<TDevice> mat_w_q(&this->weights(),                  
					     this->precedingLayer().size(), this->size(),
					     tmp_mat_size * 1);
	    // mat_q: W_q^T * x
            helpers::Matrix<TDevice> mat_q(&m_mat_q, this->size(), frame_num_total);
            mat_q.assignProduct(mat_w_q, true, mat_pre_o, false);
	    
	    // scale W_q / sequence_length
	    thrust::transform(m_mat_q.begin(), m_mat_q.end(), 
			      thrust::make_constant_iterator(sqrt(1.0/this->size())),
			      m_mat_q.begin(), thrust::multiplies<real_t>());

	    // Key
	    // mat_w_k: transformation matrix W_k
	    helpers::Matrix<TDevice> mat_w_k(&this->weights(),                  
					     this->precedingLayer().size(), this->size(),
					     tmp_mat_size * 2);
	    // mat_k: W_k ^ T * x
            helpers::Matrix<TDevice> mat_k(&m_mat_k, this->size(), frame_num_total);
            mat_k.assignProduct(mat_w_k, true, mat_pre_o, false);

	    

	    // step2. calculate the alignment matrix
	    if (this->parallelSequences() == 1){

		// Q^T * K 
		helpers::Matrix<TDevice> mat_align(&m_align, frame_num_total, frame_num_total);
		mat_align.assignProduct(mat_q, true, mat_k, false);

		
		// 1. calculate the offset to center the activations for safe exponentiation
		{{
		    internal::CalculateOffsetFn fn;
		    fn.layerSize = frame_num_total;
		    fn.outputs   = helpers::getRawPointer(m_align);

		    thrust::transform(
			thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(0) + frame_num_total,
			m_softmax_buf.begin(),
			fn);
		}}

		// 2. calculate the exponent exp(align_ij - offset + prior)
		{{
		    internal::CalculateExpFn fn;
		    fn.layerSize = frame_num_total;
		    fn.offsets   = helpers::getRawPointer(m_softmax_buf);
		    fn.prior_w   = m_align_prior_w;
		    fn.epoch     = this->getCurrTrainingEpoch();
			
		    int n = frame_num_total * frame_num_total;

		    thrust::for_each(
                     thrust::make_zip_iterator(
		      thrust::make_tuple(m_align.begin(),
					 thrust::counting_iterator<int>(0))),
		     thrust::make_zip_iterator(
		      thrust::make_tuple(m_align.begin()+n,
					 thrust::counting_iterator<int>(0)+n)),
		     fn);
		}}

		// 3. sum up all outputs for each pattern \sum_i exp(align_ij - offset)
		{{
	        helpers::Matrix<TDevice> mat_one(&m_one_vector, 1, frame_num_total);
	    	helpers::Matrix<TDevice> mat_sum(&m_softmax_buf, 1, frame_num_total);
		helpers::Matrix<TDevice> mat_align(&m_align, frame_num_total, frame_num_total);
		mat_sum.assignProduct(mat_one, false, mat_align, false);
		}}

		// 4. normalize the outputs exp(align_ij - offset) / \sum_i exp(align_ij - offset)
		{{
		    internal::NormalizeOutputsFn fn;
		    fn.layerSize = frame_num_total;
		    fn.normFacts = helpers::getRawPointer(m_softmax_buf);

		    int n = frame_num_total * frame_num_total;

		    thrust::for_each(
                     thrust::make_zip_iterator(
			thrust::make_tuple(m_align.begin(),
					   thrust::counting_iterator<int>(0))),
		     thrust::make_zip_iterator(
			thrust::make_tuple(m_align.begin()+n,
					   thrust::counting_iterator<int>(0)+n)),
		     fn);
	       }}
	    }

	    // step.3 compute output m_mat_v * align
	    // The conventional feedforward part
	    // collect outputs from preceding layer
	    {{
	        helpers::Matrix<TDevice> mat_align(&m_align, frame_num_total, frame_num_total);
	    	helpers::Matrix<TDevice> mat_v    (&m_mat_v, this->size(), frame_num_total);
		helpers::Matrix<TDevice> mat_out  (&this->_outputs(), this->size(), frame_num_total);
		mat_out.assignProduct(mat_v, false, mat_align, false);
	    }}	     
	}}
	
	// done
    }


    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::computeForwardPass(const int timeStep,
							       const int nnState)
    {
	throw std::runtime_error("Self-attention not support online generation");
    }

    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::computeBackwardPass(const int nnState)
    {
	// matrix size for W_v, W_q, and W_k
	int tmp_mat_size = this->precedingLayer().size() * this->size();

	// total number of time steps
	int frame_num_total = this->curMaxSeqLength() * this->parallelSequences();

	// step1. gradient w.r.t Q^T K
	{{
	    // gradient w.r.t alignment matrix
	    //  mat_v:          V = W_v^T * x
	    //  mat_grad_o:     \partial_E / \partial_o
	    //  mat_grad_align: V^T * \partial_E / \partial_o
	    helpers::Matrix<TDevice> mat_v   (&m_mat_v, this->size(), frame_num_total);
	    helpers::Matrix<TDevice> mat_grad_o(&this->outputErrors(),this->size(),frame_num_total);
	    helpers::Matrix<TDevice> mat_grad_align(&m_align_grad, frame_num_total, frame_num_total);
	    mat_grad_align.assignProduct(mat_v, true, mat_grad_o, false);


	    // gradient w.r.t Q^T*K (gradient propagated through softmax)
	    //  calculate the error offset for each pattern \sum_k [grad_align_kj * align_kj]
	    {{
            internal::CalculateErrorOffsetFn fn;
            fn.layerSize    = frame_num_total;
            fn.outputs      = helpers::getRawPointer(m_align);
            fn.outputErrors = helpers::getRawPointer(m_align_grad);

            thrust::transform(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(0) + frame_num_total,
                m_softmax_buf.begin(),
                fn);
	    }}
	    // calculate gradient w.r.t Q^T * K
	    //  \partial_E / \partial [Q^T*K]_ij =
	    //     align_ij * (grad_align_ij - \sum_k [grad_align_kj * align_kj] )
	    {{
            internal::CalculateErrorsFn fn;
            fn.layerSize    = frame_num_total;
            fn.errorOffsets = helpers::getRawPointer(m_softmax_buf);

            int n = frame_num_total * frame_num_total;

            thrust::for_each(
                thrust::make_zip_iterator(
			thrust::make_tuple(m_align_grad.begin(),
					   m_align.begin(),
					   thrust::counting_iterator<int>(0))),
                thrust::make_zip_iterator(
			thrust::make_tuple(m_align_grad.begin()+n,
					   m_align.begin()+n,
					   thrust::counting_iterator<int>(0)+n)),
                fn);
	    }}
	}}

	// clean the gradient buffer
	thrust::fill(this->_weightUpdates().begin(), this->_weightUpdates().end(), 0.0);
	
	// step2. gradient w.r.t W_v, x for W_v
	{{
	    // gradient w.r.t v
	    //   \parital_E / \partial_v = \partial_E / \partial_o * align^T
	    // gradient w.r.t W_v
	    //   \partial_E / \partial W_v = [\partial_E / \partial_v * x^T]^T
	    //                             = x * [\partial_E / \partial_v]^T
	    //                             = x * align * [\partial_E / \partial_o] ^ T
	    // gradient w.r.t x that propagated through v = W_v ^ T * x
	    //   \partial_E / \partial_x = W_v * \partial_E / \partial_v
	    //
	      
	    // \partial_E / \partial_v = \partial_E / \partial_o * align^T
	    //   mat_grad_o: \partial_E / \partial_o
	    //   mat_align:  align
	    //   mat_grad_buf:  \partial_E / \partial_v
	    helpers::Matrix<TDevice> mat_grad_o(&this->outputErrors(),this->size(),frame_num_total);
	    helpers::Matrix<TDevice> mat_align (&m_align, frame_num_total, frame_num_total);
	    helpers::Matrix<TDevice> mat_grad_buf(&m_grad_buf, this->size(), frame_num_total);
	    mat_grad_buf.assignProduct(mat_grad_o, false, mat_align, true);


	    // \partial_E / \partial W_v = x * [\partial_E / \partial_v] ^ T
	    //    mat_x: x
	    //    mat_grad_w_v = \partial_E / \partial W_v
	    helpers::Matrix<TDevice> mat_x(&this->precedingLayer().outputs(),
					   this->precedingLayer().size(),frame_num_total);
	    helpers::Matrix<TDevice> mat_grad_w_v (&this->_weightUpdates(),
						   this->precedingLayer().size(), this->size());
	    mat_grad_w_v.assignProduct(mat_x, false, mat_grad_buf, true);

	    // \partial_E / \partial_x = W_v * \partial_E / \partial_v 
	    //    mat_w_v:    W_v
	    //    mat_grad_x: \partial_E / \partial_x
	    helpers::Matrix<TDevice> mat_w_v(&this->weights(),                  
					     this->precedingLayer().size(), this->size());
	    helpers::Matrix<TDevice> mat_grad_x (&this->precedingLayer().outputErrors(),
						   this->precedingLayer().size(), frame_num_total);
	    mat_grad_x.assignProduct(mat_w_v, false, mat_grad_buf, false);

	    // gradoemt w.r.t q
	    //  \partial_E / \partial_q = [\partial_E / \partial_[Q^T*K] * K^T] ^ T
	    //                          = l * [\partial_E / \partial_[Q^T*K]]^T
	    // gradient w.r.t W_q
	    //  \partial_E / \partial_w_q = x * \partial_E / \partial_q ^ T
	    //  
	    // gradient w.r.t x from q
	    //  \partial_E / \partial_x = w_q * \partial_E / \partial_q
	    
	    // \partial_E / \partial_q
	    //   mat_align_grad: \partial_E / \partial_[Q^T*K]
	    //   mat_k:          k
	    //   mat_grad_buf:   \partial_E / \partial_q
	    helpers::Matrix<TDevice> mat_align_grad(&m_align_grad, frame_num_total, frame_num_total);
	    helpers::Matrix<TDevice> mat_k   (&m_mat_k, this->size(), frame_num_total);
	    mat_grad_buf.assignProduct(mat_k, false, mat_align_grad, true);
	    thrust::transform(m_grad_buf.begin(), m_grad_buf.end(), 
			      thrust::make_constant_iterator(sqrt(1.0/this->size())),
			      m_grad_buf.begin(), thrust::multiplies<real_t>());

	    // \partial_E / \partial_w_q
	    helpers::Matrix<TDevice> mat_grad_w_q (&this->_weightUpdates(),
						   this->precedingLayer().size(), this->size(),
						   tmp_mat_size * 1);
	    mat_grad_w_q.assignProduct(mat_x, false, mat_grad_buf, true);

	    // \partial_E / \partial_x
	    //   mat_w_q: w_q
	    helpers::Matrix<TDevice> mat_w_q(&this->weights(),                  
					     this->precedingLayer().size(), this->size(),
					     tmp_mat_size * 1);
	    mat_grad_x.addProduct(mat_w_q, false, mat_grad_buf, false);


	    // gradoemt w.r.t k
	    //  \partial_E / \partial_k = 
	    //                          = q * \partial_E / \partial_[Q^T*K]
	    // gradient w.r.t W_k
	    //  \partial_E / \partial_w_k = x * [\partial_E / \partial_k] ^ T
	    //  
	    // gradient w.r.t x from k
	    //  \partial_E / \partial_x = w_k * \partial_E / \partial_k
	    
	    // \partial_E / \partial_k
	    //   mat_align_grad: \partial_E / \partial_[Q^T*K]
	    //   mat_q:          q
	    //   mat_grad_buf:   \partial_E / \partial_q
	    helpers::Matrix<TDevice> mat_q   (&m_mat_q, this->size(), frame_num_total);
	    mat_grad_buf.assignProduct(mat_q, false, mat_align_grad, false);

	    // \partial_E / \partial_w_k
	    helpers::Matrix<TDevice> mat_grad_w_k (&this->_weightUpdates(),
						   this->precedingLayer().size(), this->size(),
						   tmp_mat_size * 1);
	    mat_grad_w_k.assignProduct(mat_x, false, mat_grad_buf, true);

	    // \partial_E / \partial_x
	    helpers::Matrix<TDevice> mat_w_k(&this->weights(),                  
					     this->precedingLayer().size(), this->size(),
					     tmp_mat_size * 2);
	    mat_grad_x.addProduct(mat_w_k, false, mat_grad_buf, false);
	    
	}}
	
    }


    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::computeBackwardPass(const int timeStep,
							  const int nnState)
    {
	throw std::runtime_error("self-attention doesn't support online mode");
    }

    
    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::exportLayer(
	const helpers::JsonValue     &layersArray, 
	const helpers::JsonAllocator &allocator) const
    {
	
        TrainableLayer<TDevice>::exportLayer(layersArray, allocator);
	(*layersArray)[layersArray->Size() - 1].AddMember("alignPriorWeight", m_align_prior_w,
							  allocator);
    }


    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::reduceOutputBuffer()
    {
	throw std::runtime_error("self-attention doesn't support online mode");
    }
    
    template <typename TDevice>
    int SelfAttentionLayer<TDevice>::outputBufPtrBias(const int timeStepTimesParallel,
							    const int nnState)
    {
	if (this->getSaveMemoryFlag()){
	    return timeStepTimesParallel * this->size();
	}else{
	    return 0;
	}
    }	


    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::clearAllBuffers()
    {
	this->clearOutputBuffer();
	this->__clearLocalMem();
    }

    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::resizeAllBuffers(const int timeLength)
    {
	this->resizeOutputBuffer(timeLength * this->parallelSequences() * this->size());
	this->__allocateLocalMem();
    }

    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::logAllBuffers(helpers::vecPoolManager<TDevice> &vecPoolMng,
						    bool flag_add)
    {
	// for output buffer
	Layer<TDevice>::logAllBuffers(vecPoolMng, flag_add);
	// for m_mat_v, mat_k, mat_q, grad_buf
	vecPoolMng.addOrRemoveNewVec(this->size(), flag_add);
	vecPoolMng.addOrRemoveNewVec(this->size(), flag_add);
	vecPoolMng.addOrRemoveNewVec(this->size(), flag_add);
	vecPoolMng.addOrRemoveNewVec(this->size(), flag_add);
	
	// for align and align_grad
	vecPoolMng.addOrRemoveNewVec(this->maxSeqLength(), flag_add);
	vecPoolMng.addOrRemoveNewVec(this->maxSeqLength(), flag_add);	
	
	// for oneVector and m_softmax_buf
	vecPoolMng.addOrRemoveNewVec(this->size()/this->size(), flag_add);
	vecPoolMng.addOrRemoveNewVec(this->size()/this->size(), flag_add);
    }
    
    template <typename TDevice>
    void SelfAttentionLayer<TDevice>::swapAllBuffers(helpers::vecPoolManager<TDevice> &vecPoolMng,
						     bool flag_get)
    {
	Layer<TDevice>::swapAllBuffers(vecPoolMng, flag_get);
	// for m_mat_v, k, q, grad_buf
	vecPoolMng.getSwapVector(m_mat_v,
				 this->getLayerID(), this->size(), flag_get);
	vecPoolMng.getSwapVector(m_mat_k,
				 this->getLayerID(), this->size(), flag_get);
	vecPoolMng.getSwapVector(m_mat_q,
				 this->getLayerID(), this->size(), flag_get);
	vecPoolMng.getSwapVector(m_grad_buf,
				 this->getLayerID(), this->size(), flag_get);

	// for align and align_grad
	vecPoolMng.getSwapVector(m_align,
				 this->getLayerID(), this->maxSeqLength(), flag_get);
	vecPoolMng.getSwapVector(m_align_grad,
				 this->getLayerID(), this->maxSeqLength(), flag_get);

	// for oneVector
	vecPoolMng.getSwapVector(m_one_vector,
				 this->getLayerID(), 1,            flag_get);
	vecPoolMng.getSwapVector(m_softmax_buf,
				 this->getLayerID(), 1,            flag_get);

	if (flag_get)
	    thrust::fill(m_one_vector.begin(), m_one_vector.end(), 1.0);
    }

    // explicit template instantiations
    template class SelfAttentionLayer<Cpu>;
    template class SelfAttentionLayer<Gpu>;

} // namespace layers

